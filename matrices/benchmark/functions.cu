#include "functions.cuh"

void CudaEventCreate(hipEvent_t *event)
{
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to create CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventDestroy(hipEvent_t event)
{
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to destroy CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMalloc(void **devPtr, size_t size)
{
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to allocate device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaFree(void *devPtr)
{
    hipError_t err = hipFree(devPtr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to free device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to copy memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventRecord(hipEvent_t event)
{
    hipError_t err = hipEventRecord(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to record CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
void CudaEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipEventSynchronize(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipEventElapsedTime(ms, start, end);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to calculate elapsed time %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaDeviceSynchronize()
{
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize device %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: CUDA error %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int *allocate_matrix(int rows, int n)
{
    int *matrix = (int *)calloc(rows * n, sizeof(int));
    if (matrix == NULL)
    {
        fprintf(stderr, "Error: Could not allocate memory for the matrix.\n");
        return NULL;
    }
    return matrix;
}

void generate_matrix(int rows, int cols, int *matrix)
{
    srand(time(NULL));
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            matrix[i * cols + j] = rand() % 10; // Generate numbers between 0 and 9
        }
    }
}

void print_matrix(int rows, int columns, int *matrix)
{
    for (int i = 0; i < rows; i++)
    {
        printf("| ");
        for (int j = 0; j < columns; j++)
        {
            printf("%d ", matrix[i * columns + j]);
        }
        printf("|\n");
    }
    printf("\n");
}

void fill_matrix(int rows, int columns, int *matrix, int value)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            matrix[i * columns + j] = value;
        }
    }
}
