#include "hip/hip_runtime.h"
#include "functions.cuh"

void CudaEventCreate(hipEvent_t *event)
{
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to create CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventDestroy(hipEvent_t event)
{
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to destroy CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMalloc(void **devPtr, size_t size)
{
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to allocate device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaFree(void *devPtr)
{
    hipError_t err = hipFree(devPtr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to free device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to copy memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventRecord(hipEvent_t event)
{
    hipError_t err = hipEventRecord(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to record CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
void CudaEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipEventSynchronize(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipEventElapsedTime(ms, start, end);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to calculate elapsed time %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaDeviceSynchronize()
{
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize device %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: CUDA error %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int *allocate_matrix(int rows, int n)
{
    int *matrix = (int *)calloc(rows * n, sizeof(int));
    if (matrix == NULL)
    {
        fprintf(stderr, "Error: Could not allocate memory for the matrix.\n");
        return NULL;
    }
    return matrix;
}

void generate_matrix(int rows, int cols, int *matrix)
{
    srand(time(NULL));
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            matrix[i * cols + j] = rand() % 10; // Generate numbers between 0 and 9
        }
    }
}

void print_matrix(int rows, int columns, int *matrix)
{
    for (int i = 0; i < rows; i++)
    {
        printf("| ");
        for (int j = 0; j < columns; j++)
        {
            printf("%d ", matrix[i * columns + j]);
        }
        printf("|\n");
    }
    printf("\n");
}

void fill_matrix(int rows, int columns, int *matrix, int value)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < columns; j++)
        {
            matrix[i * columns + j] = value;
        }
    }
}

void init_matrices(int matrix_size, int **A, int **B, int **C)
{
    *A = allocate_matrix(matrix_size, matrix_size);
    *B = allocate_matrix(matrix_size, matrix_size);
    *C = allocate_matrix(matrix_size, matrix_size);

    if (*A == NULL || *B == NULL || *C == NULL)
    {
        fprintf(stderr, "Error: Could not allocate memory for the matrices.\n");
        exit(EXIT_FAILURE);
    }

    generate_matrix(matrix_size, matrix_size, *A);
    generate_matrix(matrix_size, matrix_size, *B);
    fill_matrix(matrix_size, matrix_size, *C, 0);
}


void cuda_malloc_and_copy(int **d_A, int **d_B, int **d_C, int *A, int *B, int *C, int matrix_size)
{
    CudaMalloc((void **)d_A, matrix_size * matrix_size * sizeof(int));
    CudaMalloc((void **)d_B, matrix_size * matrix_size * sizeof(int));
    CudaMalloc((void **)d_C, matrix_size * matrix_size * sizeof(int));

    CudaMemcpy(*d_A, A, matrix_size * matrix_size * sizeof(int), hipMemcpyHostToDevice);
    CudaMemcpy(*d_B, B, matrix_size * matrix_size * sizeof(int), hipMemcpyHostToDevice);

    if (*d_A == NULL || *d_B == NULL || *d_C == NULL)
    {
        fprintf(stderr, "Error: Could not allocate memory on the device.\n");
        exit(EXIT_FAILURE);
    }
}

void cuda_free_matrices(int *d_A, int *d_B, int *d_C)
{
    CudaFree(d_A);
    CudaFree(d_B);
    CudaFree(d_C);
}

void free_matrices(int *A, int *B, int *C)
{
    free(A);
    free(B);
    free(C);
}

__global__ void mul(int *A, int *B, int *C, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int sum = 0;
        for (int k = 0; k < cols; k++)
        {
            sum += A[row * cols + k] * B[k * cols + col];
        }
        C[row * cols + col] = sum;
    }
}
