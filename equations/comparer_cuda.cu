#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>

#include "functions.h"
#include "functions_cuda.cuh"
#include "gauss_jordan_cuda.cuh"

double *solve_equation_with_gpu(unsigned int size, unsigned int threads, double *matrix, double *sol)
{
    double *d_matrix;
    hipEvent_t start, end;
    float ms;

    // Allocate memory on the GPU
    size_t total_size = size * (size + 1) * sizeof(double);
    CudaMalloc((void **)&d_matrix, total_size);
    CudaMemcpy(d_matrix, matrix, total_size, hipMemcpyHostToDevice);

    // Define dim3 grid dimensions
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    CudaEventCreate(&start);
    CudaEventCreate(&end);

    CudaEventRecord(start);
    // TODO: Call the kernel function
    gauss_jordan<<<numBlocks, threadsPerBlock>>>(size, d_matrix);

    CudaEventRecord(end);
    CudaEventSynchronize(end);
    CudaEventElapsedTime(&ms, start, end);

    CudaFree(d_matrix);
    CudaEventDestroy(start);
    CudaEventDestroy(end);

    for (unsigned int i = 0; i < size; i++)
        sol[i] = *(matrix + i * (size + 1) + size);

    return sol;
}

int main(int argc, char **argv)
{
    std::vector<unsigned int> sizes = {
        2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048, 4096};
    const unsigned int iterations = 32;

    double *matrix, *original_matrix = NULL, *sol, *d_matrix;
    clock_t start, end;
    double seconds;
    unsigned int total_iterations = 0;

    bool was_valid = false;

    // Generate random matrix
    srand(time(NULL));

    std::cout << "SIZE;ARCH;THRS;TIME;SUCC" << std::endl;
    for (auto size : sizes)
    {
        const unsigned int half_size = size / 2;
        for (unsigned int threads = 1; threads <= half_size; threads++)
        {
            seconds = 0;
            total_iterations = 0;
            for (unsigned int iteration = 0; iteration < iterations; iteration++)
            {
                was_valid = false;
                while (!was_valid)
                {
                    total_iterations++;
                    // Allocate memory
                    matrix = allocate_matrix(size);
                    sol = (double *)malloc(size * sizeof(double));
                    generate_matrix(size, matrix);
                    original_matrix = copy_matrix(size, matrix, original_matrix);

                    start = clock();
                    solve_equation_with_gpu(size, threads, matrix, sol);
                    end = clock();

                    // The solution is in the last column
                    for (unsigned int i = 0; i < size; i++)
                        sol[i] = *(matrix + i * (size + 1) + size);

                    seconds += (double)(end - start) / CLOCKS_PER_SEC;

                    // Check if the solution is correct
                    was_valid = check_equation_system(size, matrix, sol);

                    // Free matrix
                    free(matrix);
                    free(original_matrix);
                    free(sol);
                }
            }
            const double success_rate = (double)iterations / total_iterations;
            std::cout << size << ";" << "GPU" << ";" << threads << ";" << seconds / iterations << ";" << success_rate << std::endl;
        }
    }
}