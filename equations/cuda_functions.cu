#include "cuda_functions.cuh"

/**
 * hipEventCreate function with error handling
 * @param event Pointer to the event to be created
 */
void CudaEventCreate(hipEvent_t *event)
{
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to create CUDA event\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipEventDestroy function with error handling
 * @param event Event to be destroyed
 */
void CudaEventDestroy(hipEvent_t event)
{
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to destroy CUDA event\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipMalloc function with error handling
 * @param devPtr Pointer to the device memory
 * @param size Size of the memory to be allocated
 */
void CudaMalloc(void **devPtr, size_t size)
{
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to allocate device memory\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipFree function with error handling
 * @param devPtr Pointer to the device memory to be freed
 */
void CudaFree(void *devPtr)
{
    hipError_t err = hipFree(devPtr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to free device memory\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipMemcpy function with error handling
 * @param dst Destination pointer
 * @param src Source pointer
 * @param count Size of the memory to be copied
 * @param kind Type of memory copy (Host to Device, Device to Host, etc.)
 */
void CudaMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to copy memory\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipEventRecord function with error handling
 * @param event Event to be recorded
 * @param stream Stream in which the event is recorded
 */
void CudaEventRecord(hipEvent_t event, hipStream_t stream)
{
    hipError_t err = hipEventRecord(event, stream);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to record CUDA event\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipEventSynchronize function with error handling
 * @param event Event to be synchronized
 */
void CudaEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipEventSynchronize(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize CUDA event\n");
        exit(EXIT_FAILURE);
    }
}

/**
 * hipEventElapsedTime function with error handling
 * @param ms Pointer to store the elapsed time in milliseconds
 * @param start Start event
 * @param end End event
 */
void CudaEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipEventElapsedTime(ms, start, end);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to calculate elapsed time\n");
        exit(EXIT_FAILURE);
    }
}

