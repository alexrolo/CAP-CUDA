#include "functions_cuda.cuh"

void CudaEventCreate(hipEvent_t *event)
{
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to create CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventDestroy(hipEvent_t event)
{
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to destroy CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMalloc(void **devPtr, size_t size)
{
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to allocate device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaFree(void *devPtr)
{
    hipError_t err = hipFree(devPtr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to free device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to copy memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventRecord(hipEvent_t event)
{
    hipError_t err = hipEventRecord(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to record CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
void CudaEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipEventSynchronize(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipEventElapsedTime(ms, start, end);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to calculate elapsed time %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaDeviceSynchronize()
{
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize device %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: CUDA error %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
