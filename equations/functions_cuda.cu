#include "hip/hip_runtime.h"
#include "functions_cuda.cuh"

void CudaEventCreate(hipEvent_t *event)
{
    hipError_t err = hipEventCreate(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to create CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventDestroy(hipEvent_t event)
{
    hipError_t err = hipEventDestroy(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to destroy CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMalloc(void **devPtr, size_t size)
{
    hipError_t err = hipMalloc(devPtr, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to allocate device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaFree(void *devPtr)
{
    hipError_t err = hipFree(devPtr);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to free device memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind)
{
    hipError_t err = hipMemcpy(dst, src, count, kind);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to copy memory %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventRecord(hipEvent_t event)
{
    hipError_t err = hipEventRecord(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to record CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
void CudaEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipEventSynchronize(event);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize CUDA event %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipEventElapsedTime(ms, start, end);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to calculate elapsed time %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void CudaDeviceSynchronize()
{
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: Unable to synchronize device %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkCudaError()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Error: CUDA error %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

double *allocate_matrix(unsigned int size)
{
    double *matrix = (double *)malloc(size * size * sizeof(double *));
    if (matrix == NULL)
    {
        fprintf(stderr, "Error: Unable to allocate memory for rows.\n");
        exit(EXIT_FAILURE);
    }

    return matrix;
}

void generate_matrix(unsigned int size, double *matrix)
{
    const unsigned int total_size = size * size;
    for (unsigned int i = 0; i < total_size; i++)
        *(matrix + i) = (rand() % 10) + 1; // Generate numbers between 0 and 9
}

void print_equation_system(unsigned int size, double *matrix)
{
    const unsigned int total_size = size * size;
    for (unsigned int i = 0; i < total_size; i++)
    {
        printf("%5.4f x%d ", *(matrix + i), i % size);
        if (i % size < size - 1)
            printf("+ ");
        if (i % size == size - 1)
            printf("\n");
    }
    printf("\n");
}

int check_equation_system(unsigned int size, double *matrix, double *solution)
{
    const unsigned int total_size = size * size;
    double sum = 0;
    for (unsigned int i = 0; i < total_size; i++)
    {
        // Reset sum for each row
        if (i % size == 0)
            sum = 0;

        // Calculate the sum of the products
        sum += *(matrix + i) * *(solution + size + (i % size) * size);

        // Check if the absolute difference is small
        if (fabs(sum - *(matrix + i * size + size)) > 1e-6)
            return 0; // Solution is incorrect
    }
    return 1; // Solution is correct
}

void copy_matrix(unsigned int size, double *src, double *dest)
{
    dest = allocate_matrix(size);
    for (unsigned int i = 0; i < size; i++)
        *(dest+i) = *(src+i);
}

