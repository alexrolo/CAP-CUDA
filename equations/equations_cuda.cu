#include "hip/hip_runtime.h"
#include "functions_cuda.cuh"
#include "gauss_jordan_cuda.cuh"

double *solve_equation_with_gpu(unsigned int size, unsigned int threads, double *matrix, double *sol)
{
    double *d_matrix;
    hipEvent_t start, end;
    float ms;

    // Allocate memory on the GPU
    size_t total_size = size * (size + 1) * sizeof(double);
    CudaMalloc((void **)&d_matrix, total_size);
    CudaMemcpy(d_matrix, matrix, total_size, hipMemcpyHostToDevice);

    // Define dim3 grid dimensions
    dim3 threadsPerBlock(threads, threads);
    dim3 numBlocks((size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (size + threadsPerBlock.y - 1) / threadsPerBlock.y);

    CudaEventCreate(&start);
    CudaEventCreate(&end);

    CudaEventRecord(start);
    // TODO: Call the kernel function
    gauss_jordan<<<numBlocks, threadsPerBlock>>>(size, d_matrix);

    CudaEventRecord(end);
    CudaEventSynchronize(end);
    CudaEventElapsedTime(&ms, start, end);

    CudaFree(d_matrix);
    CudaEventDestroy(start);
    CudaEventDestroy(end);

    for (unsigned int i = 0; i < size; i++)
        sol[i] = *(matrix + i * (size + 1) + size);

    return sol;
}

int main(int argc, char *argv[])
{
    unsigned int size, threads; // Square matrix
    double *matrix, *original_matrix = NULL, *sol;
    clock_t start, end;
    double seconds;

    // Check if the required arguments are provided
    if (argc != 3)
    {
        fprintf(stderr, "Usage: %s <size> <threads_per_block>\n", argv[0]);
        return 1;
    }

    if ((size = atoi(argv[1])) < 1)
    {
        fprintf(stderr, "Invalid size: %s\n", argv[1]);
        return 1;
    }

    if ((threads = atoi(argv[2])) < 1)
    {
        fprintf(stderr, "Invalid threads per block: %s\n", argv[2]);
        return 1;
    }

    // Allocate memory
    matrix = allocate_matrix(size);
    sol = (double *)malloc(size * sizeof(double));

    // Generate random matrix
    srand(time(NULL));
    generate_matrix(size, matrix);
    original_matrix = copy_matrix(size, matrix, original_matrix);

    unsigned int i;

    start = clock();
    solve_equation_with_gpu(size, threads, matrix, sol);
    end = clock();

    // The solution is in the last column
    for (i = 0; i < size; i++)
        *(sol + i) = *(matrix + i * (size + 1) + size);

    seconds = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time (seconds): %.5f\n", seconds);

    // The solution is in the last column
    if (size < 16)
    {
        printf("System solution:\n");
        for (unsigned int i = 0; i < size; i++)
            printf("x%d = %.3f\n", i, *(sol + i));
    } else 
    {
        printf("System solution is too large to print.\n");
    }

    printf("Checking against original matrix:\n");
    if (size < 16)
        print_equation_system(size, matrix);
    else
        printf("Matrix is too large to print.\n");

    // Check if the solution is correct
    if (check_equation_system(size, original_matrix, sol))
        printf("The solution is correct.\n");
    else
        printf("The solution is incorrect.\n");

    // Free matrix
    free(matrix);
    free(original_matrix);
    free(sol);

    return 0;
}
